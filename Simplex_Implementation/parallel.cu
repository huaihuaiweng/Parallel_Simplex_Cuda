#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fstream>
#include <vector>
#include <iostream>
#include <string>
#include <sstream>

double* tableau_gpu;


struct Compare_Max {
    double val = 0;
    int index = -1;
};

int main(int argc, char** argv) {
    int nRow, nCol;
    std::ifstream tableau_file(argv[1]);
    std::cout << "argv[1]: " << argv[1] << std::endl;

    // Getting the row/col lengths
    std::string line;
    std::getline(tableau_file, line);
    std::stringstream ss(line);
    std::string curr;
    ss >> curr;
    nRow = std::stoi(curr);
    ss >> curr;
    nCol = std::stoi(curr);
    ss >> curr;

    // Moving matrix from file to vector
    std::vector<std::vector<double>> tableau_cpu(nRow, std::vector<double>(nCol, 0.0));
    for (int i = 0; i < nRow; ++i) {
        std::getline(tableau_file, line);
        std::stringstream ss(line);
        for (int j = 0; j < nCol; ++j) {
            ss >> curr;
            tableau_cpu[i][j] = std::stod(curr);
        }
    }

    // Now, print vector to check that it is loaded into memory.
    std::cout << "{";
    for (const auto& row: tableau_cpu) {
        std::cout << "{";
        for (const double& num: row) {
            std::cout << num << " ";
        }
        std::cout << "}";
        std::cout << std::endl;
    }
    std::cout << "}";

    hipMalloc((void**)&tableau_gpu, nRow * nCol * sizeof(double));
    for (int i = 0; i < nRow; ++i) {
        
    }
    hipMemcpy(tableau_gpu, tableau_cpu.)





    Compare_Max max = findMaxObjective(tableau, nRow, nCol);
    
}


__global__ void findMaxObjectiveKernel(double *tableau, int nRow, int nCol, Compare_Max *max) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (j <= nCol) {
        double val = tableau[nRow * nCol + j];
        if (val < 0.0) {
            val = -val;
            atomicMax(&(max->val), val);
            if (max->val == val) {
                max->index = j;
            }
        }
    }
}

Compare_Max findMaxObjective(double **tableau, int nRow, int nCol) {
    Compare_Max max;
    max.val = 0.0;
    max.index = -1;

    // Flatten the 2D array
    double *flat_tableau = new double[nRow * nCol];
    for (int i = 0; i < nRow; ++i) {
        for (int j = 0; j < nCol; ++j) {
            flat_tableau[i * nCol + j] = tableau[i][j];
        }
    }

    // Allocate device memory
    double *d_tableau;
    Compare_Max *d_max;
    hipMalloc((void**)&d_tableau, nRow * nCol * sizeof(double));
    hipMalloc((void**)&d_max, sizeof(Compare_Max));

    // Copy data to device
    hipMemcpy(d_tableau, flat_tableau, nRow * nCol * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_max, &max, sizeof(Compare_Max), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockSize(256);
    dim3 gridSize((nCol + blockSize.x - 1) / blockSize.x);
    findMaxObjectiveKernel<<<gridSize, blockSize>>>(d_tableau, nRow, nCol, d_max);

    // Copy result back to host
    hipMemcpy(&max, d_max, sizeof(Compare_Max), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_tableau);
    hipFree(d_max);

    // Free host memory
    delete[] flat_tableau;

    // For debugging
    cout << "max.index: " << max.index << endl;

    return max;
}
